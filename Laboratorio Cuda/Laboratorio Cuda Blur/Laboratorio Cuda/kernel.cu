#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#define BLUR_SIZE 10
__global__
void cudaBlur(unsigned char* R, unsigned char* G, unsigned char* B, unsigned char* Rout, unsigned char* Gout, unsigned char* Bout, int w,int h) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < w && row < h) {
        int Rval = 0;
        int Gval = 0;
        int Bval = 0;
        int cont = 0;
        for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; blurRow++) {
            for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; blurCol++) {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                if (curRow > -1 && curRow<h && curCol>-1 && curCol < w) {
                    Rval += R[curRow * w + curCol];
                    Gval += G[curRow * w + curCol];
                    Bval += B[curRow * w + curCol];
                    ++cont;
                }
            }
        }
        Rout[row * w + col] = (cont) ? (char)(Rval / cont) : 0;
        Gout[row * w + col] = (cont) ? (char)(Gval / cont) : 0;
        Bout[row * w + col] = (cont) ? (char)(Bval / cont) : 0;
    }
}

void blur(unsigned char* R, unsigned char* G, unsigned char* B, unsigned char* Rout, unsigned char* Gout, unsigned char* Bout, int w, int h) {
    int size = w*h * sizeof(unsigned char);
    unsigned char* d_R, * d_G, * d_B, * d_Rout, * d_Gout, * d_Bout;
    hipMalloc((void**)&d_R, size);
    hipMemcpy(d_R, R, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_G, size);
    hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_Rout, size);
    hipMalloc((void**)&d_Gout, size);
    hipMalloc((void**)&d_Bout, size);

    cudaBlur << <dim3(w/32, h/32), dim3(32, 32) >> > (d_R, d_G, d_B, d_Rout, d_Gout, d_Bout, w,h);
    hipMemcpy(Rout, d_Rout, size, hipMemcpyDeviceToHost);
    hipMemcpy(Gout, d_Gout, size, hipMemcpyDeviceToHost);
    hipMemcpy(Bout, d_Bout, size, hipMemcpyDeviceToHost);

    hipFree(d_R);
    hipFree(d_G);
    hipFree(d_B);
    hipFree(d_Rout);
    hipFree(d_Gout);
    hipFree(d_Bout);
}